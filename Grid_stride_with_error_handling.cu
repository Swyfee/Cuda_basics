
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}


void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}



__global__
void doubleElements(int *a, int N)
{
  int indexWithinTheGrid;
  indexWithinTheGrid = blockIdx.x * blockDim.x + threadIdx.x;
  int gridStride = gridDim.x * blockDim.x;
  for (int i = indexWithinTheGrid; i < N; i += gridStride)
  {
  if (i < N)
      {
        a[i] *= 2;
      }
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  /*
   * `N` is greater than the size of the grid (see below).
   */

  int N = 10000;
  int *a;

  size_t size = N * sizeof(int);
  hipError_t err;
  err =hipMallocManaged(&a, size);
if (err != hipSuccess)
{
  printf("Error: %s\n", hipGetErrorString(err));
}
  
  
  

  init(a, N);

  /*
   * The size of this grid is 256*32 = 8192.
   */

  size_t threads_per_block = 256;
  size_t number_of_blocks = 32;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
    err = hipGetLastError(); 
    if (err != hipSuccess)
    {
          printf("Error: %s\n", hipGetErrorString(err));
    }

  checkCuda( hipDeviceSynchronize() );

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}
